#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
#include <stdio.h>
__global__ void add(int a, int b, int *c, int *d){
	int bid = blockIdx.x;
	int tid = blockDim.x;
	*c = bid;
	*d = tid;
}

int main(void){
	int c,d;
	int *dev_c, *dev_d;
	hipMalloc( (void**)&dev_c, sizeof(int) );
	hipMalloc( (void**)&dev_d, sizeof(int) );	
	add<<<5,100>>>(2,7,dev_c, dev_d) ;
	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &d, dev_d, sizeof(int), hipMemcpyDeviceToHost);
	printf("this is blockIdx.x : %d\n",c);
	printf("this is blockDim.x:  %d\n",d);
	hipFree(dev_c);
	hipFree(dev_d);
	return 0; 
}
