#include "hip/hip_runtime.h"
//Program for squaring numbers in  an array on NVIOIA device
#include <stdio.h>
#include "book.h"
__global__ void square(int* a, int N)
{

}

int  main (){
	
	int* array_host,* array_device;
	int N = 10;
	array_host = (int*)malloc(N*sizeof(int));
	hipMalloc((void**)&array_device, N*sizeof(int));
	int i;
	for(i=0;i<N;i++)
	{
		array_host[i] = i;
		printf("array_host:%d \n ",array_host[i]);
	}
	int size = N;
	hipMemcpy(array_device, array_host,size,hipMemcpyHostToDevice);
	//block size and dimentions
	int blocksize = 4;
	int num_of_blocks = N/blocksize + (N%blocksize == 0 ? 0:1);
	//Kernell(deice call)
	square<<<num_of_blocks, blocksize>>>(array_device,N);
}

