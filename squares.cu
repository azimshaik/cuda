#include "hip/hip_runtime.h"
//Program for squaring numbers in  an array on NVIOIA device
#include <stdio.h>
#include "book.h"
#include <math.h>
__global__ void square(int* a, int N)
{
	//calculate the unique thread index
	//blockIdx * blockDim + threadIdx
	int tId = blockIdx.x * blockDim.x + threadIdx.x;

	if(tId<N) a[tId] = a[tId] * a[tId];
}

int  main (){
	
	int* array_host,* array_device;
	int N = 10;
	array_host = (int*)malloc(N*sizeof(int));
	hipMalloc((void**)&array_device, N*sizeof(int));
	int i;
	for(i=0;i<N;i++)
	{
		array_host[i] = i;
		printf("array_host:%d \n ",array_host[i]);
	}
	int size = N*sizeof(int);
	hipMemcpy(array_device, array_host,size,hipMemcpyHostToDevice);
	//block size and dimentions
	int blocksize = 4;
	int num_of_blocks = N/blocksize + (N%blocksize == 0 ? 0:1);
	int k;
        for (k = 0 ; k<N ; k++)
        {
                printf("The square of %d is %d \n",k, array_host[k]);;
        }
	//Kernell(deice call)
	square<<<num_of_blocks, blocksize>>>(array_device,N);
	//copy back the results from the device to host 
	hipMemcpy(array_host,array_device,sizeof(int)*N, hipMemcpyDeviceToHost);
	int j;
	for (j = 0 ; j<N ; j++)
	{
		printf("The square of %d is %d \n",j, array_host[j]);;
	}
	free(array_host);
	hipFree(array_device);
}

