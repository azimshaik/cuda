//Program for squaring numbers in  an array on NVIOIA device
#include <stdio.h>
#include "book.h"

int  main (){
	
	int* array_host,* array_device;
	int N = 10;
	array_host = (int*)malloc(N*sizeof(int));
	hipMalloc((void**)&array_device, N*sizeof(int));
	int i;
	for(i=0;i<N;i++)
	{
		array_host[i] = i;
		printf("array_host:%d \n ",array_host[i]);
	}
	int size = N;
	hipMemcpy(array_device, array_host,size,hipMemcpyHostToDevice);
	//block size and dimentions
	int blocksize = 4;
	int num_of_blocks = N/blocksize + (N%blocksize == 0 ? 0:1);
	//Kernell(deice call)

}

