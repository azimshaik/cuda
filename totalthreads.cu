#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
#include <stdio.h>
__global__ void add(int a, int b, int *c, int *d, int *e, int *f){
	int blocksPerGrid   = gridDim.x;
	int threadsPerBlock = blockDim.x;
	int totalThreadNum  = gridDim.x * blockDim.x;;
	int curThreadIdx    = (blockIdx.x * blockDim.x) + threadIdx.x;
	*c = blocksPerGrid;
	*d = threadsPerBlock;
	*e = totalThreadNum;
	*f = curThreadIdx;
}

int main(void){
	int c,d,e,f;
	int *dev_c, *dev_d, *dev_e, *dev_f;
	hipMalloc( (void**)&dev_c, sizeof(int) );
	hipMalloc( (void**)&dev_d, sizeof(int) );
	hipMalloc( (void**)&dev_e, sizeof(int) );
	hipMalloc( (void**)&dev_f, sizeof(int) );	
	add<<<5,100>>>(2,7,dev_c, dev_d, dev_e, dev_f) ;
	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &d, dev_d, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &e, dev_e, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &f, dev_f, sizeof(int), hipMemcpyDeviceToHost);
	printf("blocks Per Grid : %d\n",c);
	printf("threads per block:  %d\n",d);
	printf("total number of threads:  %d\n",e);
	printf("Current Thread  Index:  %d\n",f);
	hipFree(dev_c);
	hipFree(dev_d);
	hipFree(dev_e);
	hipFree(dev_f);
	return 0; 
}
