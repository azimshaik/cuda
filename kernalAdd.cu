#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
#include <stdio.h>
__global__ void add(int a, int b, int *c, int *d){
	int bid = threadIdx.x;
	int tid = threadIdx.y;
	*c = bid;
	*d = tid;
}

int main(void){
	int c,d;
	int *dev_c, *dev_d;
	hipMalloc( (void**)&dev_c, sizeof(int) );
	hipMalloc( (void**)&dev_d, sizeof(int) );	
	add<<<5,100>>>(2,7,dev_c, dev_d) ;
	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &d, dev_d, sizeof(int), hipMemcpyDeviceToHost);
	printf("this is threadIdx.x : %d\n",c);
	printf("this is threadIdx.x:  %d\n",d);
	hipFree(dev_c);
	hipFree(dev_d);
	return 0; 
}
