#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
#include <stdio.h>
__global__ void add(int a, int b, int *c){
	int tid = blockIdx.x +threadIdx.x;
	*c = tid;
}

int main(void){
	int c;
	int *dev_c;
	hipMalloc( (void**)&dev_c, sizeof(int) );
	
	add<<<1,10>>>(2,7,dev_c) ;
	HANDLE_ERROR(  hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
	printf("2+7= %d\n",c);
	hipFree(dev_c);
	return 0; 
}
